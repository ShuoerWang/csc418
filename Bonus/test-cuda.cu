#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <hiprand/hiprand_kernel.h>

#include <cstdio>
#include <algorithm>
#include <climits>
#include <chrono>

#include "hip/hip_vector_types.h"

#define SPP 65536
#define DEPTH 8
#define WIDTH 800
#define HEIGHT 600
#define SPHERE_EPSILON 0.0001f
#define BOX_EPSILON 0.001f
#define RAY_EPSILON 0.05f
#define M_PI 3.1415925

//main source used: 
//http://www.kevinbeason.com/smallpt/
//http://raytracey.blogspot.com/2015/10/gpu-path-tracing-tutorial-1-drawing.html

struct Ray
{
    __device__ Ray(float3 origin, float3 direction) :
        origin(origin), direction(direction) {}

    float3 origin;
    float3 direction;
};

enum Material { DIFFUSE, SPECULAR, REFRACTIVE };

struct Sphere
{
    float radius;
    float3 position, emissionColor, mainColor;
    Material material;
    __device__ float intersect(const Ray& ray) const
    {
        float t;
        float3 dis = position - ray.origin;
        float proj = dot(dis, ray.direction);
        float delta = proj * proj - dot(dis, dis) + radius * radius;
        if (delta < 0) return 0;
        delta = sqrtf(delta);
        return (t = proj - delta) > SPHERE_EPSILON ? t : ((t = proj + delta) > SPHERE_EPSILON ? t : 0);
    }
};
//https://github.com/matt77hias/cu-smallpt/blob/master/cu-smallpt/cu-smallpt/src/kernel.cu
__constant__ Sphere spheres[] =
{

    //{ 1e5f, { -1e5f - 50.0f, 40.0f, 80.0f }, { 0.0f, 0.0f, 0.0f }, { 0.75f, 0.25f, 0.25f }, Material::DIFFUSE }, // Left 
    //{ 1e5f, { 1e5f + 50.0f, 40.0f, 80.0f }, { 0.0f, 0.0f, 0.0f }, { 0.25f, 0.25f, 0.75f }, Material::DIFFUSE }, // Right 
    //{ 1e5f, { 0.0f, 40.0f, -1e5f }, { 0.0f, 0.0f, 0.0f }, { 0.75f, 0.75f, 0.75f }, Material::DIFFUSE }, // Back 
    //{ 1e5f, { 0.0f, 40.0f, 1e5f + 600.0f }, { 0.0f, 0.0f, 0.0f }, { 1.00f, 1.00f, 1.00f }, Material::DIFFUSE }, // Front 
    //{ 1e5f, { 0.0f, -1e5f, 80.0f }, { 0.0f, 0.0f, 0.0f }, { 0.75f, 0.75f, 0.75f }, Material::DIFFUSE }, // Bottom 
    //{ 1e5f, { 0.0f, 1e5f + 80.0f, 80.0f }, { 0.0f, 0.0f, 0.0f }, { 0.75f, 0.75f, 0.75f }, Material::DIFFUSE }, // Top 
    //{ 16.0f, { -25.0f, 16.0f, 47.0f }, { 0.0f, 0.0f, 0.0f }, { 2.0f, 2.0f, 2.0f }, Material::REFRACTIVE }, // left sphere 
    //{ 20.0f, { 25.0f, 20.0f, 78.0f }, { 0.0f, 0.0f, 0.0f }, { 1.0f, 1.0f, 1.0f }, Material::SPECULAR }, // right sphere
    ////{ 5.0f, { 35.0f, 74.0f, 40.0f }, { 5.6f, 5.6f, 5.6f }, { 0.0f, 0.0f, 0.0f }, Material::DIFFUSE }  // Light
    //{ 600.0f, { 0.0f, 678.8f, 80.0f }, { 1.6f, 1.6f, 1.6f }, { 0.0f, 0.0f, 0.0f }, Material::DIFFUSE }  // Light

    //{ 1e5f, { -1e5f - 50.0f, 40.0f, 80.0f }, { 0.0f, 0.0f, 0.0f }, { 0.6f, 0.2f, 0.2f }, Material::DIFFUSE }, // Left 
    //{ 1e5f, { 1e5f + 50.0f, 40.0f, 80.0f }, { 0.0f, 0.0f, 0.0f }, { 0.2f, 0.2f, 0.6f }, Material::DIFFUSE }, // Right 
    //{ 1e5f, { 0.0f, 40.0f, -1e5f }, { 0.0f, 0.0f, 0.0f }, { 0.75f, 0.75f, 0.75f }, Material::DIFFUSE }, // Back 
    //{ 1e5f, { 0.0f, 40.0f, 1e5f + 600.0f }, { 0.0f, 0.0f, 0.0f }, { 1.00f, 1.00f, 1.00f }, Material::DIFFUSE }, // Front 
    //{ 1e5f, { 0.0f, -1e5f, 80.0f }, { 0.0f, 0.0f, 0.0f }, { 0.75f, 0.75f, 0.75f }, Material::DIFFUSE }, // Bottom 
    //{ 1e5f, { 0.0f, 1e5f + 80.0f, 80.0f }, { 0.0f, 0.0f, 0.0f }, { 0.75f, 0.75f, 0.75f }, Material::DIFFUSE }, // Top \
    ////sphere
    //{ 12.0f, { -30.0f, 12.0f, 88.0f }, { 0.0f, 0.0f, 0.0f }, { 1.0f, 1.0f, 1.0f }, Material::REFRACTIVE },
    //{ 15.0f, { 25.0f, 15.0f, 47.0f }, { 0.0f, 0.0f, 0.0f }, { 0.65f, 0.65f, 0.65f }, Material::SPECULAR }, 
    //{ 8.0f, { 10.0f, 8.0f, 130.0f }, { 0.0f, 0.0f, 0.0f }, { 1.0f, 1.0f, 1.0f }, Material::REFRACTIVE },
    //{ 12.0f, { 25.0f, 12.0f, 180.0f }, { 0.0f, 0.0f, 0.0f }, { 0.3f, 0.8f, 0.3f }, Material::DIFFUSE }, 
    //{ 10.0f, { -15.0f, 10.0f, 200.0f }, { 0.0f, 0.0f, 0.0f }, { 0.2f, 0.9f, 0.9f }, Material::SPECULAR }, 
    //{ 6.0f, { 0.0f, 6.0f, 230.0f }, { 0.0f, 0.0f, 0.0f }, { 0.9f, 0.9f, 0.2f }, Material::REFRACTIVE },

    ////light
    //{ 5.0f, { 35.0f, 74.0f, 40.0f }, { 15.6f, 15.6f, 15.6f }, { 0.0f, 0.0f, 0.0f }, Material::DIFFUSE }, 
    //{ 5.0f, { 35.0f, 74.0f, 140.0f }, { 15.6f, 15.6f, 15.6f }, { 0.0f, 0.0f, 0.0f }, Material::DIFFUSE },
    //{ 5.0f, { 35.0f, 74.0f, 240.0f }, { 15.6f, 15.6f, 15.6f }, { 0.0f, 0.0f, 0.0f }, Material::DIFFUSE }, 
    //{ 5.0f, { 35.0f, 74.0f, 340.0f }, { 15.6f, 15.6f, 15.6f }, { 0.0f, 0.0f, 0.0f }, Material::DIFFUSE }, 
    
    
    { 1e5f, { 0.0f, -1e5f, 80.0f }, { 0.0f, 0.0f, 0.0f }, { 0.3f, 0.3f, 0.3f }, Material::DIFFUSE },// ground
    { 4e4f, { 0.0f, -4e4f-30, -3000.0f }, { 0.0f, 0.0f, 0.0f }, { 0.2f, 0.2f, 0.2f }, Material::DIFFUSE },// mountain
    { 110000.0f, { 0.0f, -110050.0f, 80.0f }, { 3.6f, 2.0f, 00.2f }, { 0.0f, 0.0f, 0.0f }, Material::DIFFUSE }, //horizon
    {1600.0f, {2000.0f, 200.0f, 8000.0f }, {37.44f, 33.696f, 29.952f} , { 0.0f, 0.0f, 0.0f }, DIFFUSE},// sun
    {1560.0f, {2000.0f, 200.0f, 8000.0f }, {150.0f, 75.0f, 7.488f} , { 0.0f, 0.0f, 0.0f }, DIFFUSE},//sun
    {10000.0f, {0.0f, 0.0f, -1060.0f}, {0.000306f, 0.01f, 0.1388315664f}, {0.175f, 0.175f , 0.25f},  DIFFUSE}, // sky

    //sphere
    
    /*{ 18.0f, { -30.0f, 18.0f, -42.0f }, { 0.0f, 0.0f, 0.0f }, { 0.8f, 0.8f, 0.8f }, Material::REFRACTIVE },
    { 25.0f, { 25.0f, 25.0f, -83.0f }, { 0.0f, 0.0f, 0.0f }, { 0.65f, 0.65f, 0.65f }, Material::SPECULAR }, 
    { 8.0f, { 10.0f, 8.0f, 0.0f }, { 0.0f, 0.0f, 0.0f }, { 0.92f, 0.92f, 0.92f }, Material::REFRACTIVE },
    { 12.0f, { 25.0f, 12.0f, 50.0f }, { 0.0f, 0.0f, 0.0f }, { 0.2f, 0.2f, 0.6f }, Material::REFRACTIVE },
    { 10.0f, { -15.0f, 10.0f, 70.0f }, { 0.0f, 0.0f, 0.0f }, { 0.2f, 0.9f, 0.9f }, Material::SPECULAR }, 
    { 6.0f, { 0.0f, 6.0f, 90.0f }, { 0.0f, 0.0f, 0.0f }, { 0.8f, 0.8f, 0.2f }, Material::REFRACTIVE },*/
    { 18.0f, { -30.0f, 18.0f, 0.0f }, { 0.0f, 0.0f, 0.0f }, { 0.2f, 0.2f, 0.6f }, Material::REFRACTIVE },
    { 25.0f, { 25.0f, 25.0f, -20.0f }, { 0.0f, 0.0f, 0.0f }, { 0.65f, 0.65f, 0.65f }, Material::SPECULAR },
    { 12.0f, { -10.0f, 12.0f, 60.0f }, { 0.0f, 0.0f, 0.0f }, { 0.7f, 0.7f, 0.7f }, Material::REFRACTIVE },

    //{1600.0f, {3000.0f, 0.0f, 6000.0f }, {37.44f, 33.696f, 29.952f} , { 0.0f, 0.0f, 0.0f }, DIFFUSE}, //sun
    //{10000.0f, {50.0f, 40.8f, -1060.0f}, {0.000306f, 0.01f, 0.1388315664f}, {0.175f, 0.175f , 0.25f},  DIFFUSE}, // sky
    //{100000.0f, {50.0f, -100000.0f, 0.0f}, { 0.0f, 0.0f, 0.0f } ,{0.3f, 0.3f, 0.3f}, DIFFUSE },
    //{110000.0f, {50.0f, -100048.5f, 0.0f}, { 0.0f, 0.0f, 0.0f } ,{3.6f, 2.0f, 00.2f}, DIFFUSE },

    //{ 13.0f, { -75.0f, 13.0f, 282.0f }, { 0.0f, 0.0f, 0.0f }, { 0.2f, 0.9f, 0.9f }, Material::SPECULAR }, 

};

__device__ inline bool hit(const Ray& ray, float& t, int& id)
{
    t = FLT_MAX, id = -1;
    int sphereNum = sizeof(spheres) / sizeof(Sphere);
    for (int i = 0; i < sphereNum; i++)
    {
        float ct = spheres[i].intersect(ray);
        if (ct != 0 && ct < t)
        {
            t = ct;
            id = i;
        }
    }

    return id != -1;
}


inline int gammaCorrect(float c)
{
    return int(pow(clamp(c, 0.0f, 1.0f), 1 / 2.2) * 255 + .5);
}

__device__ float3 radiance(Ray& ray, hiprandState* rs)
{
    float3 result = make_float3(0.0f, 0.0f, 0.0f);
    float3 temp = make_float3(1.0f, 1.0f, 1.0f);

    for (int i = 0; i < DEPTH; i++)
    {
        //http://raytracey.blogspot.com/2015/10/gpu-path-tracing-tutorial-1-drawing.html
        float t;
        int id;

        if (!hit(ray, t, id))
            break;

        const Sphere& obj = spheres[id];
        float3 hit_point = ray.origin + ray.direction * t;
        float3 n = normalize(hit_point - obj.position);
        float3 nl = dot(n, ray.direction) < 0 ? n : n * -1;

        ray.origin = hit_point + nl * RAY_EPSILON;

        result += temp * obj.emissionColor;

        if (obj.material == DIFFUSE) {
            float r1 = hiprand_uniform(rs) * M_PI * 2;
            float r2 = hiprand_uniform(rs);
            float r2s = sqrtf(r2);

            float3 w = nl;
            float3 u = normalize(cross((std::fabs(w.x) > std::fabs(w.y) ? make_float3(0.0f, 1.0f, 0.0f) : make_float3(1.0f, 0.0f, 0.0f)), w));
            float3 v = cross(w, u);

            ray.direction = normalize(u * cos(r1) * r2s + v * sin(r1) * r2s + w * sqrtf(1 - r2));

            temp *= obj.mainColor * dot(ray.direction, nl) * 2;
        }
        else if (obj.material == SPECULAR) {
            ray.direction = ray.direction - 2 * n * dot(n, ray.direction);
            temp *= obj.mainColor;
        }
        else {
            ray.origin = hit_point;
            float3 reflect = ray.direction - 2 * n * dot(n, ray.direction);

            float nc = 1, nt = 1.5, cos2t;
            float ddn = dot(ray.direction, nl);
            float nnt;
            if (dot(n, nl) > 0) {
                nnt = nc / nt;
            }
            else {
                nnt = nt / nc;
            }
            bool into = dot(n, nl) > 0;
            if ((cos2t = 1 - nnt * nnt * (1 - ddn * ddn)) < 0) {
                ray.direction = reflect;
                temp *= obj.mainColor;
            }
            else {
                float3 tdir = normalize(ray.direction * nnt - n * ((into ? 1 : -1) * (ddn * nnt + sqrt(cos2t))));
                float a = nt - nc, b = nt + nc, R0 = a * a / (b * b), c = 1 - (into ? -ddn : dot(tdir, n));
                float Re = R0 + (1 - R0) * c * c * c * c * c, Tr = 1 - Re, P = .25 + .5 * Re, RP = Re / P, TP = Tr / (1 - P);

                if(hiprand_uniform(rs) < P)
                {
                    ray.direction = reflect;
                    temp *= obj.mainColor * RP;
                }
                else
                {
                    ray.direction = tdir;
                    temp *= obj.mainColor * TP;
                }
            }
        }
    }

    return result;
}



__global__ void launchkernel(float3* cpu_out)
{
    uint x = blockIdx.x * blockDim.x + threadIdx.x;
    uint y = blockIdx.y * blockDim.y + threadIdx.y;

    uint i = (HEIGHT - y - 1) * WIDTH + x;

    hiprandState rs;
    hiprand_init(i, 0, 0, &rs);

    /*Ray cam(make_float3(0.0f, 52.0f, 300.0f), normalize(make_float3(0.0f, -0.05f, -1.0f)));*/
    Ray cam(make_float3(0.0f, 42.0f, 250.0f), normalize(make_float3(0.0f, -0.035f, -1.0f)));
    float3 cx = make_float3(WIDTH * 0.5135 / HEIGHT, 0.0f, 0.0f);
    float3 cy = normalize(cross(cx, cam.direction)) * 0.5135;
    float3 pixel = make_float3(0.0f);

    //http://www.kevinbeason.com/smallpt/
    for (int sy = 0; sy < 2; sy++) {
        for (int sx = 0; sx < 2; sx++) { 
            for(int s = 0; s < SPP; s++) {
                float r1 = hiprand_uniform(&rs);
                float dx = r1 < 1 ? sqrtf(r1) - 1 : 1-sqrtf(2 - r1);
                float r2 = hiprand_uniform(&rs);
                float dy = r2 < 1 ? sqrtf(r2) - 1 : 1-sqrtf(2 - r2);
                float3 d = cam.direction + cx*((((sx + dx + .5) / 2) + x) / WIDTH - .5) + cy*((((sy + dy + .5) / 2) + y) / HEIGHT - .5);
                Ray tRay(cam.origin + d * 140, normalize(d));
                tRay.direction = normalize(d);
                tRay.origin = cam.origin + d * 140;
                pixel += radiance(tRay, &rs) *(0.25f / SPP);
            }
        }
    }

    cpu_out[i] = clamp(pixel, 0.0f, 1.0f);
    
}

int main() {
    float3* cpu_out = new float3[WIDTH * HEIGHT];
    float3* gpu_out;
    hipMalloc(&gpu_out, WIDTH * HEIGHT * sizeof(float3));
    dim3 block(32, 32);
    dim3 grid(WIDTH / block.x, HEIGHT / block.y);
    launchkernel << <grid, block >> > (gpu_out);
    hipMemcpy(cpu_out, gpu_out, WIDTH * HEIGHT * sizeof(float3), hipMemcpyDeviceToHost);
    hipFree(gpu_out);
    FILE* f = fopen("cudayes3.ppm", "w");
    fprintf(f, "P3\n%d %d\n%d\n", WIDTH, HEIGHT, 255);
    for (int i = 0; i < WIDTH * HEIGHT; i++)
        fprintf(f, "%d %d %d ", gammaCorrect(cpu_out[i].x),  gammaCorrect(cpu_out[i].y), gammaCorrect(cpu_out[i].z));
    delete[] cpu_out;

    return 0;
}